#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"


const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************

void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  *acc = new int[rBins * degreeBins];
  memset (*acc, 0, sizeof (int) * rBins * degreeBins);
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) 
    for (int j = 0; j < h; j++) 
      {
        int idx = j * w + i;
        if (pic[idx] > 0) 
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  
            float theta = 0;         
            for (int tIdx = 0; tIdx < degreeBins; tIdx++)
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++;
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// inicializarlo en main y pasarlo al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

// The accummulator memory needs to be allocated by the host in global memory
// __global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x ;
  if (gloID > w * h) return;

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);
	hipEventRecord(start);
  // GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);
  // ----------------------------------------
  //
  // THIS IS THE PARALLEL BLOCK
  //
  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);
  // ----------------------------------------
  hipEventRecord(stop);
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");
  printf("Time: %f\n", milliseconds);

  // clean-up
	hipFree((void *) d_Cos);
	hipFree((void *) d_Sin);
	hipFree((void *) d_in);
	hipFree((void *) d_hough);
	free(pcCos);
	free(pcSin);
	free(h_hough);

  return 0;
}
